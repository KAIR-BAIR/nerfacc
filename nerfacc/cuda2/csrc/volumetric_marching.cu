
#include <hip/hip_runtime.h>
// #include <pybind11/pybind11.h>
// #include "include/helpers_cuda.h"

// // Perform fixed-size stepping in unit-cube scenes (like original NeRF) and exponential
// // stepping in larger scenes.
// inline CUDA_HOSTDEV float calc_dt(float t, float cone_angle, float dt_min, float dt_max)
// {
//     return __clamp(t * cone_angle, dt_min, dt_max);
// }

// inline CUDA_HOSTDEV int cascaded_grid_idx_at(
//     const float x, const float y, const float z,
//     const int resx, const int resy, const int resz)
// {
//     // TODO(ruilongli): if the x, y, z is outside the aabb, it will be clipped into aabb!!! We should just return false
//     int ix = (int)(x * resx);
//     int iy = (int)(y * resy);
//     int iz = (int)(z * resz);
//     ix = __clamp(ix, 0, resx - 1);
//     iy = __clamp(iy, 0, resy - 1);
//     iz = __clamp(iz, 0, resz - 1);
//     int idx = ix * resy * resz + iy * resz + iz;
//     return idx;
// }

// inline CUDA_HOSTDEV bool normalize_with_contraction(
//     float x, float y, float z,
//     const float *aabb, 
//     const int contraction_type, 
//     const bool normalize, // If true, it will output normalized coordinates in [0, 1]
//     float *outx, float *outy, float *outz
// ){
//     // normalize and contract to a unit space.
//     switch (contraction_type)
//     {
//     case 0:
//         // no contraction
//         if (normalize) {
//             *outx = (x - aabb[0]) / (aabb[3] - aabb[0]);
//             *outy = (y - aabb[1]) / (aabb[4] - aabb[1]);
//             *outz = (z - aabb[2]) / (aabb[5] - aabb[2]);
//         }
//         else {
//             *outx = x;
//             *outy = y;
//             *outz = z;
//         }
//         break;
//     case 1:
//         // mipnerf360 scene contraction
//         // The aabb defines a sphere in which the samples are not modified. 
//         // The samples outside the sphere are contracted into a 2x radius sphere.
//         x = (x - aabb[0]) / (aabb[3] - aabb[0]) * 2.0f - 1.0f;
//         y = (y - aabb[1]) / (aabb[4] - aabb[1]) * 2.0f - 1.0f;
//         z = (z - aabb[2]) / (aabb[5] - aabb[2]) * 2.0f - 1.0f;
//         float norm = sqrt(x * x + y * y + z * z);
//         if (norm > 1.0f)
//         {
//             x = (2.0f - 1.0f / norm) * (x / norm);
//             y = (2.0f - 1.0f / norm) * (y / norm);
//             z = (2.0f - 1.0f / norm) * (z / norm);
//         }
//         x = (x * 0.5f + 1.0f) * 0.5f; // the first 0.5f is bc of the 2x radius
//         y = (y * 0.5f + 1.0f) * 0.5f;
//         z = (z * 0.5f + 1.0f) * 0.5f;
//         if (normalize) {
//             *outx = x;
//             *outy = y;
//             *outz = z;
//         }
//         else {
//             *outx = 2.0f * (x - 0.25f) * (aabb[3] - aabb[0]) + aabb[0];
//             *outy = 2.0f * (y - 0.25f) * (aabb[4] - aabb[1]) + aabb[1];
//             *outz = 2.0f * (z - 0.25f) * (aabb[5] - aabb[2]) + aabb[2];
//         }
//         break;
//     }
// }

// inline CUDA_HOSTDEV bool grid_occupied_at(
//     float x, float y, float z,
//     const int resx, const int resy, const int resz,
//     const float *aabb, const bool *occ_binary, const int contraction_type)
// {
//     // normalize and maybe contract the coordinates.
//     float _x, _y, _z;
//     normalize_with_contraction(
//         x, y, z, aabb, contraction_type, true, &_x, &_y, &_z);
//     int idx = cascaded_grid_idx_at(_x, _y, _z, resx, resy, resz);
//     return occ_binary[idx];
// }

// inline CUDA_HOSTDEV float distance_to_next_voxel(
//     float x, float y, float z,
//     float dir_x, float dir_y, float dir_z,
//     float idir_x, float idir_y, float idir_z,
//     const int resx, const int resy, const int resz,
//     const float *aabb)
// { // dda like step
//     // TODO: this is ugly -- optimize this.
//     float _x = ((x - aabb[0]) / (aabb[3] - aabb[0])) * resx;
//     float _y = ((y - aabb[1]) / (aabb[4] - aabb[1])) * resy;
//     float _z = ((z - aabb[2]) / (aabb[5] - aabb[2])) * resz;
//     float tx = ((floorf(_x + 0.5f + 0.5f * __sign(dir_x)) - _x) * idir_x) / resx * (aabb[3] - aabb[0]);
//     float ty = ((floorf(_y + 0.5f + 0.5f * __sign(dir_y)) - _y) * idir_y) / resy * (aabb[4] - aabb[1]);
//     float tz = ((floorf(_z + 0.5f + 0.5f * __sign(dir_z)) - _z) * idir_z) / resz * (aabb[5] - aabb[2]);
//     float t = min(min(tx, ty), tz);
//     return fmaxf(t, 0.0f);
// }

// inline CUDA_HOSTDEV float advance_to_next_voxel(
//     float t,
//     float x, float y, float z,
//     float dir_x, float dir_y, float dir_z,
//     float idir_x, float idir_y, float idir_z,
//     const int resx, const int resy, const int resz, const float *aabb,
//     float dt_min)
// {
//     // Regular stepping (may be slower but matches non-empty space)
//     float t_target = t + distance_to_next_voxel(
//                              x, y, z,
//                              dir_x, dir_y, dir_z,
//                              idir_x, idir_y, idir_z,
//                              resx, resy, resz, aabb);
//     do
//     {
//         t += dt_min;
//     } while (t < t_target);
//     return t;
// }

// __global__ void marching_steps_kernel(
//     // rays info
//     const uint32_t n_rays,
//     const float *rays_o, // shape (n_rays, 3)
//     const float *rays_d, // shape (n_rays, 3)
//     const float *t_min,  // shape (n_rays,)
//     const float *t_max,  // shape (n_rays,)
//     // density grid
//     const float *aabb, // [min_x, min_y, min_z, max_x, max_y, max_z]
//     const int resx,
//     const int resy,
//     const int resz,
//     const bool *occ_binary, // shape (reso_x, reso_y, reso_z)
//     // sampling
//     const float step_size,
//     const int contraction_type,
//     const float cone_angle,
//     // outputs
//     int *num_steps)
// {
//     CUDA_GET_THREAD_ID(i, n_rays);

//     // locate
//     rays_o += i * 3;
//     rays_d += i * 3;
//     t_min += i;
//     t_max += i;
//     num_steps += i;

//     const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
//     const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
//     const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
//     const float near = t_min[0], far = t_max[0];

//     float dt_min = step_size;
//     float dt_max = 1e10f; // TODO: if not contraction, calculate from occ res and aabb

//     int j = 0;
//     float t0 = near;
//     float dt = calc_dt(t0, cone_angle, dt_min, dt_max);
//     float t1 = t0 + dt;
//     float t_mid = (t0 + t1) * 0.5f;

//     while (t_mid < far)
//     {
//         // current center
//         const float x = ox + t_mid * dx;
//         const float y = oy + t_mid * dy;
//         const float z = oz + t_mid * dz;

//         if (grid_occupied_at(x, y, z, resx, resy, resz, aabb, occ_binary, contraction_type))
//         {
//             ++j;
//             // march to next sample
//             t0 = t1;
//             t1 = t0 + calc_dt(t0, cone_angle, dt_min, dt_max);
//             t_mid = (t0 + t1) * 0.5f;
//         }
//         else
//         {
//             // march to next sample
//             switch (contraction_type)
//             {
//             case 0:
//                 // no contraction
//                 t_mid = advance_to_next_voxel(
//                     t_mid, x, y, z, dx, dy, dz, rdx, rdy, rdz, resx, resy, resz, aabb, dt_min);
//                 dt = calc_dt(t_mid, cone_angle, dt_min, dt_max);
//                 t0 = t_mid - dt * 0.5f;
//                 t1 = t_mid + dt * 0.5f;
//                 break;
            
//             default:
//                 // any type of scene contraction does not work with DDA.
//                 t0 = t1;
//                 t1 = t0 + calc_dt(t0, cone_angle, dt_min, dt_max);
//                 t_mid = (t0 + t1) * 0.5f;
//                 break;
//             }
//         }
//     }
//     if (j == 0)
//         return;

//     num_steps[0] = j;
//     return;
// }

// __global__ void marching_forward_kernel(
//     // rays info
//     const uint32_t n_rays,
//     const float *rays_o, // shape (n_rays, 3)
//     const float *rays_d, // shape (n_rays, 3)
//     const float *t_min,  // shape (n_rays,)
//     const float *t_max,  // shape (n_rays,)
//     // density grid
//     const float *aabb, // [min_x, min_y, min_z, max_x, max_y, max_y]
//     const int resx,
//     const int resy,
//     const int resz,
//     const bool *occ_binary, // shape (reso_x, reso_y, reso_z)
//     // sampling
//     const float step_size,
//     const int contraction_type,
//     const float cone_angle,
//     const int *packed_info,
//     // frustrum outputs
//     float *frustum_starts,
//     float *frustum_ends)
// {
//     CUDA_GET_THREAD_ID(i, n_rays);

//     // locate
//     rays_o += i * 3;
//     rays_d += i * 3;
//     t_min += i;
//     t_max += i;
//     int base = packed_info[i * 2 + 0];
//     int steps = packed_info[i * 2 + 1];

//     const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
//     const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
//     const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
//     const float near = t_min[0], far = t_max[0];

//     // locate
//     frustum_starts += base;
//     frustum_ends += base;

//     float dt_min = step_size;
//     float dt_max = 1e10f; // TODO: if not contraction, calculate from occ res and aabb

//     int j = 0;
//     float t0 = near;
//     float dt = calc_dt(t0, cone_angle, dt_min, dt_max);
//     float t1 = t0 + dt;
//     float t_mid = (t0 + t1) * 0.5f;

//     while (t_mid < far)
//     {
//         // current center
//         const float x = ox + t_mid * dx;
//         const float y = oy + t_mid * dy;
//         const float z = oz + t_mid * dz;

//         if (grid_occupied_at(x, y, z, resx, resy, resz, aabb, occ_binary, contraction_type))
//         {
//             frustum_starts[j] = t0;
//             frustum_ends[j] = t1;
//             ++j;
//             // march to next sample
//             t0 = t1;
//             t1 = t0 + calc_dt(t0, cone_angle, dt_min, dt_max);
//             t_mid = (t0 + t1) * 0.5f;
//         }
//         else
//         {
//             // march to next sample
//             switch (contraction_type)
//             {
//             case 0:
//                 // no contraction
//                 t_mid = advance_to_next_voxel(
//                     t_mid, x, y, z, dx, dy, dz, rdx, rdy, rdz, resx, resy, resz, aabb, dt_min);
//                 dt = calc_dt(t_mid, cone_angle, dt_min, dt_max);
//                 t0 = t_mid - dt * 0.5f;
//                 t1 = t_mid + dt * 0.5f;
//                 break;
            
//             default:
//                 // any type of scene contraction does not work with DDA.
//                 t0 = t1;
//                 t1 = t0 + calc_dt(t0, cone_angle, dt_min, dt_max);
//                 t_mid = (t0 + t1) * 0.5f;
//                 break;
//             }
//         }
//     }

//     if (j != steps)
//     {
//         printf("WTF %d v.s. %d\n", j, steps);
//     }
//     return;
// }

// __global__ void ray_indices_kernel(
//     // input
//     const int n_rays,
//     const int *packed_info,
//     // output
//     int *ray_indices)
// {
//     CUDA_GET_THREAD_ID(i, n_rays);

//     // locate
//     const int base = packed_info[i * 2 + 0];  // point idx start.
//     const int steps = packed_info[i * 2 + 1]; // point idx shift.
//     if (steps == 0)
//         return;

//     ray_indices += base;

//     for (int j = 0; j < steps; ++j)
//     {
//         ray_indices[j] = i;
//     }
// }

// __global__ void occ_query_kernel(
//     // rays info
//     const uint32_t n_samples,
//     const float *samples, // shape (n_samples, 3)
//     // density grid
//     const float *aabb, // [min_x, min_y, min_z, max_x, max_y, max_y]
//     const int resx,
//     const int resy,
//     const int resz,
//     const bool *occ_binary, // shape (reso_x, reso_y, reso_z)
//     // sampling
//     const int contraction_type,
//     // outputs
//     bool *occs)
// {
//     CUDA_GET_THREAD_ID(i, n_samples);

//     // locate
//     samples += i * 3;
//     occs += i;

//     occs[0] = grid_occupied_at(
//         samples[0], samples[1], samples[2],
//         resx, resy, resz, aabb, occ_binary, contraction_type);
//     return;
// }

// __global__ void contraction_kernel(
//     // rays info
//     const uint32_t n_samples,
//     const float *samples, // shape (n_samples, 3)
//     // contraction
//     const float *aabb, // [min_x, min_y, min_z, max_x, max_y, max_y]
//     const int contraction_type,
//     // outputs
//     float *out_samples)
// {
//     CUDA_GET_THREAD_ID(i, n_samples);

//     // locate
//     samples += i * 3;
//     out_samples += i * 3;

//     normalize_with_contraction(
//         samples[0], samples[1], samples[2],
//         aabb, contraction_type, false, 
//         &out_samples[0], &out_samples[1], &out_samples[2]
//     );
//     return;
// }

// std::vector<torch::Tensor> volumetric_marching(
//     // rays
//     const torch::Tensor rays_o,
//     const torch::Tensor rays_d,
//     const torch::Tensor t_min,
//     const torch::Tensor t_max,
//     // density grid
//     const torch::Tensor aabb,
//     const pybind11::list resolution,
//     const torch::Tensor occ_binary,
//     // sampling
//     const float step_size,
//     const int contraction_type,
//     const float cone_angle)
// {
//     DEVICE_GUARD(rays_o);

//     CHECK_INPUT(rays_o);
//     CHECK_INPUT(rays_d);
//     CHECK_INPUT(t_min);
//     CHECK_INPUT(t_max);
//     CHECK_INPUT(aabb);
//     CHECK_INPUT(occ_binary);

//     const int n_rays = rays_o.size(0);

//     const int threads = 256;
//     const int blocks = CUDA_N_BLOCKS_NEEDED(n_rays, threads);

//     // helper counter
//     torch::Tensor num_steps = torch::zeros(
//         {n_rays}, rays_o.options().dtype(torch::kInt32));

//     // count number of samples per ray
//     marching_steps_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
//         // rays
//         n_rays,
//         rays_o.data_ptr<float>(),
//         rays_d.data_ptr<float>(),
//         t_min.data_ptr<float>(),
//         t_max.data_ptr<float>(),
//         // density grid
//         aabb.data_ptr<float>(),
//         resolution[0].cast<int>(),
//         resolution[1].cast<int>(),
//         resolution[2].cast<int>(),
//         occ_binary.data_ptr<bool>(),
//         // sampling
//         step_size,
//         contraction_type,
//         cone_angle,
//         // outputs
//         num_steps.data_ptr<int>());

//     torch::Tensor cum_steps = num_steps.cumsum(0, torch::kInt32);
//     torch::Tensor packed_info = torch::stack({cum_steps - num_steps, num_steps}, 1);
//     // std::cout << "num_steps" << num_steps.dtype() << std::endl;
//     // std::cout << "cum_steps" << cum_steps.dtype() << std::endl;
//     // std::cout << "packed_info" << packed_info.dtype() << std::endl;

//     // output frustum samples
//     int total_steps = cum_steps[cum_steps.size(0) - 1].item<int>();
//     torch::Tensor frustum_starts = torch::zeros({total_steps, 1}, rays_o.options());
//     torch::Tensor frustum_ends = torch::zeros({total_steps, 1}, rays_o.options());

//     marching_forward_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
//         // rays
//         n_rays,
//         rays_o.data_ptr<float>(),
//         rays_d.data_ptr<float>(),
//         t_min.data_ptr<float>(),
//         t_max.data_ptr<float>(),
//         // density grid
//         aabb.data_ptr<float>(),
//         resolution[0].cast<int>(),
//         resolution[1].cast<int>(),
//         resolution[2].cast<int>(),
//         occ_binary.data_ptr<bool>(),
//         // sampling
//         step_size,
//         contraction_type,
//         cone_angle,
//         packed_info.data_ptr<int>(),
//         // outputs
//         frustum_starts.data_ptr<float>(),
//         frustum_ends.data_ptr<float>());

//     return {packed_info, frustum_starts, frustum_ends};
// }

// torch::Tensor unpack_to_ray_indices(const torch::Tensor packed_info)
// {
//     DEVICE_GUARD(packed_info);
//     CHECK_INPUT(packed_info);

//     const int n_rays = packed_info.size(0);
//     const int threads = 256;
//     const int blocks = CUDA_N_BLOCKS_NEEDED(n_rays, threads);

//     int n_samples = packed_info[n_rays - 1].sum(0).item<int>();
//     torch::Tensor ray_indices = torch::zeros(
//         {n_samples}, packed_info.options().dtype(torch::kInt32));

//     ray_indices_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
//         n_rays,
//         packed_info.data_ptr<int>(),
//         ray_indices.data_ptr<int>());
//     return ray_indices;
// }

// torch::Tensor query_occ(
//     const torch::Tensor samples,
//     // density grid
//     const torch::Tensor aabb,
//     const pybind11::list resolution,
//     const torch::Tensor occ_binary,
//     // sampling
//     const int contraction_type)
// {
//     DEVICE_GUARD(samples);
//     CHECK_INPUT(samples);

//     const int n_samples = samples.size(0);
//     const int threads = 256;
//     const int blocks = CUDA_N_BLOCKS_NEEDED(n_samples, threads);

//     torch::Tensor occs = torch::zeros(
//         {n_samples}, samples.options().dtype(torch::kBool));

//     occ_query_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
//         n_samples,
//         samples.data_ptr<float>(),
//         // density grid
//         aabb.data_ptr<float>(),
//         resolution[0].cast<int>(),
//         resolution[1].cast<int>(),
//         resolution[2].cast<int>(),
//         occ_binary.data_ptr<bool>(),
//         // sampling
//         contraction_type,
//         // outputs
//         occs.data_ptr<bool>());
//     return occs;
// }

// torch::Tensor contraction(
//     const torch::Tensor samples,
//     // contraction
//     const torch::Tensor aabb,
//     const int contraction_type)
// {
//     DEVICE_GUARD(samples);
//     CHECK_INPUT(samples);

//     const int n_samples = samples.size(0);
//     const int threads = 256;
//     const int blocks = CUDA_N_BLOCKS_NEEDED(n_samples, threads);

//     torch::Tensor out_samples = torch::zeros({n_samples, 3}, samples.options());

//     contraction_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
//         n_samples,
//         samples.data_ptr<float>(),
//         // density grid
//         aabb.data_ptr<float>(),
//         contraction_type,
//         // outputs
//         out_samples.data_ptr<float>()
//     );
//     return out_samples;
// }
